#include "hip/hip_runtime.h"
/**
 * @file backward.cu
 * @author xbillowy
 * @brief 
 * @version 0.1
 * @date 2024-08-26
 * 
 * @copyright Copyright (c) 2024
 * 
 */

#define OPTIXU_MATH_DEFINE_IN_NAMESPACE

#include <optix.h>
#include <hip/hip_math_constants.h>

#include "params.h"
#include "auxiliary.h"


// Make the parameters available to the device code
extern "C" {
    __constant__ Params params;
}


// Unpack two 32-bit payload from a 64-bit pointer
static __forceinline__ __device__
void *unpackPointer(uint32_t i0, uint32_t i1) {
    const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}
// Pack a 64-bit pointer from two 32-bit payload
static __forceinline__ __device__
void packPointer(void* ptr, uint32_t& i0, uint32_t& i1) {
    const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}
// Get the payload pointer
template<typename T>
static __forceinline__ __device__ T *getPayload() {
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return reinterpret_cast<T*>(unpackPointer(u0, u1));
}
// Call optixTrace to trace a single ray
__device__ void traceStep(float3 ray_o, float3 ray_d, uint32_t payload_u0, uint32_t payload_u1)
{
    optixTrace(
        params.handle,
        ray_o,
        ray_d,
        0.0f,  // Min intersection distance
        1e16,  // Max intersection distance
        0.0f,  // rayTime, used for motion blur, disable
        OptixVisibilityMask(0xFF),
        OPTIX_RAY_FLAG_NONE,
        0,  // SBT offset
        0,  // SBT stride
        0,  // missSBTIndex
        payload_u0, payload_u1);
}

// Forward method for converting the input spherical harmonics
// coefficients of each Gaussian to a simple RGB color
__device__ glm::vec3 computeColorFromSHForward(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, float* clamped)
{
	// The implementation is loosely based on code for 
	// "Differentiable Point-Based Radiance Fields for 
	// Efficient View Synthesis" by Zhang et al. (2022)
	glm::vec3 pos = means[idx];
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;
	glm::vec3 result = SH_C0 * sh[0];

	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2)
			{
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
	result += 0.5f;

	// RGB colors are clamped to positive values. If values are
	// clamped, we need to keep track of this for the backward pass.
	clamped[0] = (result.x < 0);
	clamped[1] = (result.y < 0);
	clamped[2] = (result.z < 0);
	return glm::max(result, 0.0f);
}


// Backward pass for conversion of spherical harmonics to RGB for
// each Gaussian
__device__ void computeColorFromSHBackward(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, const float* clamped, const float* dL_dcolor, glm::vec3* dL_dmeans, glm::vec3* dL_dshs)
{
	// Compute intermediate values, as it is done during forward
	glm::vec3 pos = means[idx];
	glm::vec3 dir_orig = pos - campos;
	glm::vec3 dir = dir_orig / glm::length(dir_orig);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;

	// Use PyTorch rule for clamping: if clamping was applied,
	// gradient becomes 0.
	// glm::vec3 dL_dRGB = dL_dcolor[idx];
	glm::vec3 dL_dRGB = glm::vec3(dL_dcolor[0], dL_dcolor[1], dL_dcolor[2]);
	dL_dRGB.x *= clamped[0] ? 0 : 1;
	dL_dRGB.y *= clamped[1] ? 0 : 1;
	dL_dRGB.z *= clamped[2] ? 0 : 1;

	glm::vec3 dRGBdx(0, 0, 0);
	glm::vec3 dRGBdy(0, 0, 0);
	glm::vec3 dRGBdz(0, 0, 0);
	float x = dir.x;
	float y = dir.y;
	float z = dir.z;

	// Target location for this Gaussian to write SH gradients to
	glm::vec3* dL_dsh = dL_dshs + idx * max_coeffs;

	// No tricks here, just high school-level calculus.
	float dRGBdsh0 = SH_C0;
	glm::vec3 dL_dsh0 = dRGBdsh0 * dL_dRGB;
	atomicAdd(&(dL_dsh[0].x), dL_dsh0.x);
	atomicAdd(&(dL_dsh[0].y), dL_dsh0.y);
	atomicAdd(&(dL_dsh[0].z), dL_dsh0.z);
	if (deg > 0)
	{
		float dRGBdsh1 = -SH_C1 * y;
		float dRGBdsh2 = SH_C1 * z;
		float dRGBdsh3 = -SH_C1 * x;
		glm::vec3 dL_dsh1 = dRGBdsh1 * dL_dRGB;
		glm::vec3 dL_dsh2 = dRGBdsh2 * dL_dRGB;
		glm::vec3 dL_dsh3 = dRGBdsh3 * dL_dRGB;
		atomicAdd(&(dL_dsh[1].x), dL_dsh1.x);
		atomicAdd(&(dL_dsh[1].y), dL_dsh1.y);
		atomicAdd(&(dL_dsh[1].z), dL_dsh1.z);
		atomicAdd(&(dL_dsh[2].x), dL_dsh2.x);
		atomicAdd(&(dL_dsh[2].y), dL_dsh2.y);
		atomicAdd(&(dL_dsh[2].z), dL_dsh2.z);
		atomicAdd(&(dL_dsh[3].x), dL_dsh3.x);
		atomicAdd(&(dL_dsh[3].y), dL_dsh3.y);
		atomicAdd(&(dL_dsh[3].z), dL_dsh3.z);

		dRGBdx = -SH_C1 * sh[3];
		dRGBdy = -SH_C1 * sh[1];
		dRGBdz = SH_C1 * sh[2];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;

			float dRGBdsh4 = SH_C2[0] * xy;
			float dRGBdsh5 = SH_C2[1] * yz;
			float dRGBdsh6 = SH_C2[2] * (2.f * zz - xx - yy);
			float dRGBdsh7 = SH_C2[3] * xz;
			float dRGBdsh8 = SH_C2[4] * (xx - yy);
			glm::vec3 dL_dsh4 = dRGBdsh4 * dL_dRGB;
			glm::vec3 dL_dsh5 = dRGBdsh5 * dL_dRGB;
			glm::vec3 dL_dsh6 = dRGBdsh6 * dL_dRGB;
			glm::vec3 dL_dsh7 = dRGBdsh7 * dL_dRGB;
			glm::vec3 dL_dsh8 = dRGBdsh8 * dL_dRGB;
			atomicAdd(&(dL_dsh[4].x), dL_dsh4.x);
			atomicAdd(&(dL_dsh[4].y), dL_dsh4.y);
			atomicAdd(&(dL_dsh[4].z), dL_dsh4.z);
			atomicAdd(&(dL_dsh[5].x), dL_dsh5.x);
			atomicAdd(&(dL_dsh[5].y), dL_dsh5.y);
			atomicAdd(&(dL_dsh[5].z), dL_dsh5.z);
			atomicAdd(&(dL_dsh[6].x), dL_dsh6.x);
			atomicAdd(&(dL_dsh[6].y), dL_dsh6.y);
			atomicAdd(&(dL_dsh[6].z), dL_dsh6.z);
			atomicAdd(&(dL_dsh[7].x), dL_dsh7.x);
			atomicAdd(&(dL_dsh[7].y), dL_dsh7.y);
			atomicAdd(&(dL_dsh[7].z), dL_dsh7.z);
			atomicAdd(&(dL_dsh[8].x), dL_dsh8.x);
			atomicAdd(&(dL_dsh[8].y), dL_dsh8.y);
			atomicAdd(&(dL_dsh[8].z), dL_dsh8.z);

			dRGBdx += SH_C2[0] * y * sh[4] + SH_C2[2] * 2.f * -x * sh[6] + SH_C2[3] * z * sh[7] + SH_C2[4] * 2.f * x * sh[8];
			dRGBdy += SH_C2[0] * x * sh[4] + SH_C2[1] * z * sh[5] + SH_C2[2] * 2.f * -y * sh[6] + SH_C2[4] * 2.f * -y * sh[8];
			dRGBdz += SH_C2[1] * y * sh[5] + SH_C2[2] * 2.f * 2.f * z * sh[6] + SH_C2[3] * x * sh[7];

			if (deg > 2)
			{
				float dRGBdsh9 = SH_C3[0] * y * (3.f * xx - yy);
				float dRGBdsh10 = SH_C3[1] * xy * z;
				float dRGBdsh11 = SH_C3[2] * y * (4.f * zz - xx - yy);
				float dRGBdsh12 = SH_C3[3] * z * (2.f * zz - 3.f * xx - 3.f * yy);
				float dRGBdsh13 = SH_C3[4] * x * (4.f * zz - xx - yy);
				float dRGBdsh14 = SH_C3[5] * z * (xx - yy);
				float dRGBdsh15 = SH_C3[6] * x * (xx - 3.f * yy);
				glm::vec3 dL_dsh9 = dRGBdsh9 * dL_dRGB;
				glm::vec3 dL_dsh10 = dRGBdsh10 * dL_dRGB;
				glm::vec3 dL_dsh11 = dRGBdsh11 * dL_dRGB;
				glm::vec3 dL_dsh12 = dRGBdsh12 * dL_dRGB;
				glm::vec3 dL_dsh13 = dRGBdsh13 * dL_dRGB;
				glm::vec3 dL_dsh14 = dRGBdsh14 * dL_dRGB;
				glm::vec3 dL_dsh15 = dRGBdsh15 * dL_dRGB;
				atomicAdd(&(dL_dsh[9].x), dL_dsh9.x);
				atomicAdd(&(dL_dsh[9].y), dL_dsh9.y);
				atomicAdd(&(dL_dsh[9].z), dL_dsh9.z);
				atomicAdd(&(dL_dsh[10].x), dL_dsh10.x);
				atomicAdd(&(dL_dsh[10].y), dL_dsh10.y);
				atomicAdd(&(dL_dsh[10].z), dL_dsh10.z);
				atomicAdd(&(dL_dsh[11].x), dL_dsh11.x);
				atomicAdd(&(dL_dsh[11].y), dL_dsh11.y);
				atomicAdd(&(dL_dsh[11].z), dL_dsh11.z);
				atomicAdd(&(dL_dsh[12].x), dL_dsh12.x);
				atomicAdd(&(dL_dsh[12].y), dL_dsh12.y);
				atomicAdd(&(dL_dsh[12].z), dL_dsh12.z);
				atomicAdd(&(dL_dsh[13].x), dL_dsh13.x);
				atomicAdd(&(dL_dsh[13].y), dL_dsh13.y);
				atomicAdd(&(dL_dsh[13].z), dL_dsh13.z);
				atomicAdd(&(dL_dsh[14].x), dL_dsh14.x);
				atomicAdd(&(dL_dsh[14].y), dL_dsh14.y);
				atomicAdd(&(dL_dsh[14].z), dL_dsh14.z);
				atomicAdd(&(dL_dsh[15].x), dL_dsh15.x);
				atomicAdd(&(dL_dsh[15].y), dL_dsh15.y);
				atomicAdd(&(dL_dsh[15].z), dL_dsh15.z);

				dRGBdx += (
					SH_C3[0] * sh[9] * 3.f * 2.f * xy +
					SH_C3[1] * sh[10] * yz +
					SH_C3[2] * sh[11] * -2.f * xy +
					SH_C3[3] * sh[12] * -3.f * 2.f * xz +
					SH_C3[4] * sh[13] * (-3.f * xx + 4.f * zz - yy) +
					SH_C3[5] * sh[14] * 2.f * xz +
					SH_C3[6] * sh[15] * 3.f * (xx - yy));

				dRGBdy += (
					SH_C3[0] * sh[9] * 3.f * (xx - yy) +
					SH_C3[1] * sh[10] * xz +
					SH_C3[2] * sh[11] * (-3.f * yy + 4.f * zz - xx) +
					SH_C3[3] * sh[12] * -3.f * 2.f * yz +
					SH_C3[4] * sh[13] * -2.f * xy +
					SH_C3[5] * sh[14] * -2.f * yz +
					SH_C3[6] * sh[15] * -3.f * 2.f * xy);

				dRGBdz += (
					SH_C3[1] * sh[10] * xy +
					SH_C3[2] * sh[11] * 4.f * 2.f * yz +
					SH_C3[3] * sh[12] * 3.f * (2.f * zz - xx - yy) +
					SH_C3[4] * sh[13] * 4.f * 2.f * xz +
					SH_C3[5] * sh[14] * (xx - yy));
			}
		}
	}

	// The view direction is an input to the computation. View direction
	// is influenced by the Gaussian's mean, so SHs gradients
	// must propagate back into 3D position.
	glm::vec3 dL_ddir(glm::dot(dRGBdx, dL_dRGB), glm::dot(dRGBdy, dL_dRGB), glm::dot(dRGBdz, dL_dRGB));

	// Account for normalization of direction
	float3 dL_dmean = dnormvdv(float3{ dir_orig.x, dir_orig.y, dir_orig.z }, float3{ dL_ddir.x, dL_ddir.y, dL_ddir.z });

	// Gradients of loss w.r.t. Gaussian means, but only the portion 
	// that is caused because the mean affects the view-dependent color.
	// Additional mean gradient is accumulated in below methods.
	atomicAdd(&(dL_dmeans[idx].x), dL_dmean.x);
	atomicAdd(&(dL_dmeans[idx].y), dL_dmean.y);
	atomicAdd(&(dL_dmeans[idx].z), dL_dmean.z);
}


// Compute a 2D-to-2D mapping matrix from world to splat space,
// given a 2D gaussian parameters
__device__ void compute_transmat_uv_forward(
	const glm::vec3 p_orig,
	const glm::vec2 scale,
	float mod,
	const glm::vec4 rot,
	const float* viewmatrix,
    float3& xyz,
	float& dpt,
	const float3 ray_oc,
	const float3 ray_dc,
	glm::mat3x4& world2splat,
	float3& normal,
    float2& uv
) {
    // Convert the quaternion and scale vector to matrices
    // * NOTE: R here is the row-major rotation matrix, namely R as in Python,
    // * NOTE: the original quat_to_rotmat(rot) will return the column-major R^T
    // * NOTE: S here is the inverse of the scale matrix
	glm::mat3 R = quat_to_rotmat_transpose(rot);
	glm::mat3 S = scale_to_mat_inverse(scale, mod);
	glm::mat3 L = S * R;
    glm::vec3 T = -1.f * L * p_orig;

	// Compute the world to splat transformation matrix
	world2splat = glm::mat3x4(
        glm::vec4(L[0].x, L[1].x, L[2].x, T.x),
        glm::vec4(L[0].y, L[1].y, L[2].y, T.y),
        glm::vec4(L[0].z, L[1].z, L[2].z, T.z)
	);

    // Compute the normal in world space
	normal = make_float3(L[0].z, L[1].z, L[2].z);

	float3 mu = make_float3(p_orig.x, p_orig.y, p_orig.z);
	dpt = -sumf3((ray_oc - mu) * normal) / sumf3(ray_dc * normal);
	xyz = ray_oc + ray_dc * dpt;

    // Convert the intersection point from world to splat space
    glm::vec3 uv1 = glm::vec4(glm::vec3(xyz.x, xyz.y, xyz.z), 1.0f) * world2splat;
    uv = make_float2(uv1.x, uv1.y);
}


__device__ void compute_transmat_uv_backward(
	const glm::vec3 p_orig,
	const glm::vec2 scale, 
	float mod,
	const glm::vec4 rot,
	const float* viewmatrix,
	const float3 dir,
    const float3 xyz,
	const glm::mat3x4 world2splat,
	const float3 normal,
    const float2 uv,
	const float* dL_dnorm,
	const float2 dL_duv,
	glm::vec2& dL_dscale,
	glm::vec4& dL_drot,
	glm::vec3& dL_dmean3D)
{
    // Convert the quaternion and scale vector to matrices
    // * NOTE: R here is the row-major rotation matrix, namely R as in Python,
    // * NOTE: the original quat_to_rotmat(rot) will return the column-major R^T
    // * NOTE: S here is the inverse of the scale matrix
	glm::mat3 R = quat_to_rotmat_transpose(rot);
	glm::mat3 S = scale_to_mat_inverse(scale, mod);
	glm::mat3 L = S * R;

	// Compute the gradient w.r.t. the world2splat matrix
	glm::mat3x4 dL_dworld2splat = glm::mat3x4(
		glm::vec4(xyz.x, xyz.y, xyz.z, 1.0) * dL_duv.x,
		glm::vec4(xyz.x, xyz.y, xyz.z, 1.0) * dL_duv.y,
		glm::vec4(0.0, 0.0, 0.0, 0.0)
	);

	// Compute the gradient w.r.t. the original normal first
	float3 dL_dtw = make_float3(dL_dnorm[0], dL_dnorm[1], dL_dnorm[2]);
#if DUAL_VISIABLE
	float cos = -sumf3(dir * normal);
	dL_dtw = cos > 0 ? dL_dtw : -dL_dtw;
#endif

	// Compute the gradient w.r.t. L
	glm::mat3 dL_dL = glm::mat3(
		glm::vec3(
			dL_dworld2splat[0].x - dL_dworld2splat[0].w * p_orig.x,
			dL_dworld2splat[1].x - dL_dworld2splat[1].w * p_orig.x,
			dL_dworld2splat[2].x - dL_dworld2splat[2].w * p_orig.x + dL_dtw.x
		),
		glm::vec3(
			dL_dworld2splat[0].y - dL_dworld2splat[0].w * p_orig.y,
			dL_dworld2splat[1].y - dL_dworld2splat[1].w * p_orig.y,
			dL_dworld2splat[2].y - dL_dworld2splat[2].w * p_orig.y + dL_dtw.y
		),
		glm::vec3(
			dL_dworld2splat[0].z - dL_dworld2splat[0].w * p_orig.z,
			dL_dworld2splat[1].z - dL_dworld2splat[1].w * p_orig.z,
			dL_dworld2splat[2].z - dL_dworld2splat[2].w * p_orig.z + dL_dtw.z
		)
	);

	// Update gradient w.r.t. scale, rotation and mean3D
	glm::mat3 dL_dR = glm::mat3(
		dL_dL[0] / glm::vec3(scale, 1.f),
		dL_dL[1] / glm::vec3(scale, 1.f),
		dL_dL[2] / glm::vec3(scale, 1.f)
	);
	dL_drot = quat_to_rotmat_vjp(rot, glm::transpose(dL_dR));
	dL_dscale = glm::vec2(
		-(dL_dL[0].x * R[0].x + dL_dL[1].x * R[1].x + dL_dL[2].x * R[2].x) / scale.x / scale.x,
		-(dL_dL[0].y * R[0].y + dL_dL[1].y * R[1].y + dL_dL[2].y * R[2].y) / scale.y / scale.y
	);
	dL_dmean3D = glm::vec3(
		-(dL_dworld2splat[0].w * L[0].x + dL_dworld2splat[1].w * L[0].y + dL_dworld2splat[2].w * L[0].z),
		-(dL_dworld2splat[0].w * L[1].x + dL_dworld2splat[1].w * L[1].y + dL_dworld2splat[2].w * L[1].z),
		-(dL_dworld2splat[0].w * L[2].x + dL_dworld2splat[1].w * L[2].y + dL_dworld2splat[2].w * L[2].z)
	);
}


// Core __raygen__ program
extern "C" __global__ void __raygen__ot()
{
    // Lookup current location within the launch grid
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();
    uint32_t tidx = idx.x * dim.y + idx.y;
    // bool flag=false;
    // if (idx.x == 25 && idx.y == 1000)
    // {
	// 	printf("backward\n");
    //     flag = true;
    // }

    // Fetch the ray origin and direction of the current pixel
    float3 ray_om = params.ray_o[tidx];
    float3 ray_dm = params.ray_d[tidx];
    // Store a copy of the original ray origin and direction
    float3 ray_oc = ray_om;
    float3 ray_dc = ray_dm;

    // Creat and initialize the ray payload data
    RayPayload payload;
    IntersectionInfo buffer[CHUNK_SIZE];
    for (int i = 0; i < CHUNK_SIZE; i++) buffer[i].tmx = 1e16f;
    payload.buffer = buffer;
    payload.dpt = 0.f;
    payload.cnt = 0.f;

    // Pack the pointer, the values we store the payload pointer in
    uint32_t payload_u0, payload_u1;
    packPointer(&payload, payload_u0, payload_u1);
	
    // Prepare rendering data
	float C[3] = {0.0f};
	float clamped[3];
    float dpt = 0.0f;
	glm::mat3x4 world2splat;
    float3 xyz = make_float3(0.0f, 0.0f, 0.0f);
    float3 normal;
    float2 uv;

	float dL_drgb[3];
	dL_drgb[0] = params.dL_dout_attr_float32[NUM_CHANNELS_F * tidx + INTENSITY_OFFSET];
	dL_drgb[1] = params.dL_dout_attr_float32[NUM_CHANNELS_F * tidx + RAYHIT_OFFSET];
	dL_drgb[2] = params.dL_dout_attr_float32[NUM_CHANNELS_F * tidx + RAYDROP_OFFSET];
	float dL_ddpt = params.dL_dout_attr_float32[NUM_CHANNELS_F * tidx + DEPTH_OFFSET];
	float dL_dacc = params.dL_dout_attr_float32[NUM_CHANNELS_F * tidx + ACCUM_OFFSET];
	float dL_dnorm[3];
	for (int i = 0; i < 3; i++)
		dL_dnorm[i] = params.dL_dout_attr_float32[NUM_CHANNELS_F * tidx + NORMAL_OFFSET + i];
	const float T_final = params.out_attr_float32[NUM_CHANNELS_F * tidx + FINALT_OFFSET];
	const float final_D = params.out_attr_float32[NUM_CHANNELS_F * tidx + DISDEPTH_OFFSET];
	const float final_D2 = params.out_attr_float32[NUM_CHANNELS_F * tidx + DISDEPTH2_OFFSET];
	float dL_dmedian_depth = params.dL_dout_attr_float32[NUM_CHANNELS_F * tidx + MIDDEPTH_OFFSET];
	float dL_ddistortion = params.dL_dout_attr_float32[NUM_CHANNELS_F * tidx + DISTORTION_OFFSET];

    int* m_buffer = params.out_attr_uint32 + NUM_CHANNELS_I * tidx + M_BUFFER;
	const int median_contributor = params.out_attr_uint32[NUM_CHANNELS_I * tidx + MID_CONTRIB_OFFSET];
    const int contributor = params.out_attr_uint32[NUM_CHANNELS_I * tidx + N_CONTRIB_OFFSET];

	// TODO (xbillowy): Implement this?
	// float dL_ddist = params.dL_ddist[tidx];
	// Prepare gradients computation data
	const float final_A = 1 - T_final;
	float T = T_final;
	float last_dL_dT = 0;
	float last_color[3] = {0};
	float acc_colors[3] = {0};
	float last_depth = 0;
	float acc_depths = 0;
	float last_alpha = 0;
	float acc_alphas = 0;
	float last_normal[3] = {0};
	float acc_normals[3] = {0};
	// // What's this?
	// float last_dL_dT = 0;
	// Per-Gaussian gradient
	float dL_dcolor[3];
	glm::vec2 dL_dscale;
	glm::vec4 dL_drot;
	glm::vec3 dL_dmean3D;


    for (int i = contributor - 1; i >= 0; i--)
	{
		int gidx = m_buffer[i];


		// Build the world to splat transformation matrix
		compute_transmat_uv_forward(params.means3D[gidx], params.scales[gidx],
									params.scale_modifier, params.rotations[gidx], params.viewmatrix,
									xyz, dpt, ray_oc, ray_dc, world2splat, normal, uv);

		// Get weights
		float rho3d = uv.x * uv.x + uv.y * uv.y;
		float rho2d = rho3d;
		// Get particle response
		float power = -0.5f * min(rho3d, rho2d);
		if (power > 0.0f)
			continue;

		// Eq. (2) from 3D Gaussian splatting paper
		// Obtain alpha by multiplying with Gaussian opacity
		// and its exponential falloff from mean
		const float G = exp(power);
		float alpha = min(0.99f, params.opacities[gidx] * G);
		if (alpha < 1.0f / 255.0f)
			continue;

		T = T / (1.f - alpha);
		const float dchannel_dcolor = alpha * T;
		// TODO (xbillowy): What's this?
		// const float w = alpha * T;
		// if (flag)
		//     printf("gidx: %d\n", gidx);


		// Compute or fetch forward color or feature first
		if (params.colors_precomp == nullptr)
		{
			glm::vec3 result = computeColorFromSHForward(gidx, params.D, params.M,
														params.means3D, *params.campos,
														params.shs, clamped);
			C[0] = result.x;
			C[1] = result.y;
			C[2] = result.z;
		}
		const float* feature_ptr = params.colors_precomp != nullptr ? params.colors_precomp : C;
		// Propagate gradients to per-Gaussian colors and keep
		// gradients w.r.t. alpha (blending factor for a Gaussian/pixel pair)
		float dL_dalpha = 0.0f;
		for (int ch = 0; ch < 3; ch++)
		{
			const float c = feature_ptr[ch];
			acc_colors[ch] = last_alpha * last_color[ch] + (1.f - last_alpha) * acc_colors[ch];
			// Update last color (to be used in the next iteration)
			last_color[ch] = c;

			const float dL_dchannel = dL_drgb[ch];
			dL_dalpha += (c - acc_colors[ch]) * dL_dchannel;
			dL_dcolor[ch] = dchannel_dcolor * dL_dchannel;
			// Update the gradients w.r.t. color of the Gaussian.
			// Atomic, since this pixel is just one of potentially
			// many that were affected by this Gaussian
			atomicAdd(&(params.dL_dcolors[ch + 3 * gidx]), dchannel_dcolor * dL_dchannel);
		}

		float dL_dt = 0.0f;
		float dL_dnormal_gs[3] = {0.0f};
#if RENDER_AXUTILITY
		// Propagate gradients w.r.t mid T depths
		if (gidx == median_contributor) {
			dL_dt += dL_dmedian_depth;
		}

		// propagate the current weight W_{i} to next weight W_{i-1}
		const float m_d = dpt;
		const float dmd_dd = 1;
#if DETACH_WEIGHT 
		// if not detached weight, sometimes will bia toward creating extragated 2D Gaussians near front
		float dL_dweight = 0;
#else
		float dL_dweight = (final_D2 + m_d * m_d * final_A - 2 * m_d * final_D) * dL_ddistortion;
#endif
		dL_dalpha += dL_dweight - last_dL_dT;
		last_dL_dT = dL_dweight * alpha + (1 - alpha) * last_dL_dT;
		const float dL_dmd = 2.0f * (T * alpha) * (m_d * final_A - final_D) * dL_ddistortion;
		dL_dt += dL_dmd * dmd_dd;

		// Propagate gradients w.r.t. color ray-splat alphas
		acc_alphas = last_alpha * 1.0 + (1.f - last_alpha) * acc_alphas;
		dL_dalpha += (1 - acc_alphas) * dL_dacc;

		// Propagate gradients w.r.t ray-splat depths
		acc_depths = last_alpha * last_depth + (1.f - last_alpha) * acc_depths;
		last_depth = dpt;
		dL_dalpha += (dpt - acc_depths) * dL_ddpt;
		dL_dt += alpha * T * dL_ddpt; 

		// Propagate gradients to per-Gaussian normals
		float normal_tmp[3] = {normal.x, normal.y, normal.z};
		for (int ch = 0; ch < 3; ch++) {
			acc_normals[ch] = last_alpha * last_normal[ch] + (1.f - last_alpha) * acc_normals[ch];
			last_normal[ch] = normal_tmp[ch];
			dL_dalpha += (normal_tmp[ch] - acc_normals[ch]) * dL_dnorm[ch];
			dL_dnormal_gs[ch] = alpha * T * dL_dnorm[ch];
		}
#endif
		// Update dL_dalpha of current Gaussian
		dL_dalpha *= T;
		// Update last alpha (to be used in the next iteration)
		last_alpha = alpha;

		// Account for fact that alpha also influences how much of
		// the background color is added if nothing left to blend
		float bg_x_drgb = 0;
		for (int ch = 0; ch < 3; ch++)
			bg_x_drgb += params.background[ch] * dL_drgb[ch];
		dL_dalpha += (-T_final / (1.f - alpha)) * bg_x_drgb;

		// Helpful reusable temporary variables
		const float dL_dG = params.opacities[gidx] * dL_dalpha;
		// float dL_dz = 0.0f;
		// dL_dz += alpha * T * dL_ddpt;

		// Update gradients w.r.t. covariance of Gaussian 3x3 (T)
		const float2 dL_duv = {dL_dG * -G * uv.x, dL_dG * -G * uv.y};

		// Update gradients w.r.t. opacity of the Gaussian
		atomicAdd(&(params.dL_dopacities[gidx]), G * dL_dalpha);
		
		// Compute gradients w.r.t. scaling, rotation, position of the Gaussian
#if DUAL_VISIABLE
		float3 dir = make_float3(params.means3D[gidx].x - ray_o_forward.x, params.means3D[gidx].y - ray_o_forward.y, params.means3D[gidx].z - ray_o_forward.z);
		// float3 dir = ray_dm;
#endif
		compute_transmat_uv_backward(params.means3D[gidx], params.scales[gidx],
									params.scale_modifier, params.rotations[gidx], params.viewmatrix,
									dir, xyz, world2splat, normal, uv, dL_dnormal_gs, dL_duv,
									dL_dscale, dL_drot, dL_dmean3D);
		// Update gradients w.r.t. scaling
		atomicAdd(&(params.dL_dscales[gidx].x), dL_dscale.x);
		atomicAdd(&(params.dL_dscales[gidx].y), dL_dscale.y);
		// Update gradients w.r.t. rotation
		atomicAdd(&(params.dL_drotations[gidx].x), dL_drot.x);
		atomicAdd(&(params.dL_drotations[gidx].y), dL_drot.y);
		atomicAdd(&(params.dL_drotations[gidx].z), dL_drot.z);
		atomicAdd(&(params.dL_drotations[gidx].w), dL_drot.w);
		// Update gradients w.r.t. position of the Gaussian
		atomicAdd(&(params.dL_dmeans3D[gidx].x), dL_dmean3D.x);
		atomicAdd(&(params.dL_dmeans3D[gidx].y), dL_dmean3D.y);
		atomicAdd(&(params.dL_dmeans3D[gidx].z), dL_dmean3D.z);

		// Compute the gradient w.r.t. the SHs if they are present
		if (params.colors_precomp == nullptr)
			computeColorFromSHBackward(gidx, params.D, params.M, params.means3D, *params.campos,
									params.shs, clamped, dL_dcolor,
									params.dL_dmeans3D, params.dL_dshs);	
	}
}

// Core __anyhit__ program
extern "C" __global__ void __anyhit__ot()
{
    // https://forums.developer.nvidia.com/t/some-confusion-on-anyhit-shader-in-optix/223336
    // Get the payload pointer
    RayPayload &payload = *getPayload<RayPayload>();

    // Get the intersection tmax and the primitive index
    float tmx = optixGetRayTmax();
    uint32_t idx = optixGetPrimitiveIndex();

    // Increment the number of intersections
    if (tmx < payload.buffer[CHUNK_SIZE - 1].tmx)
    {
        // Enter this branch means current intersection is closer, we need to update the buffer
        // Increment the counter, the counter only increases when the intersection is closer
        payload.cnt += 1;

        // Temporary variable for swapping
        float tmp_tmx;
        float cur_tmx = tmx;
        uint32_t tmp_idx;
        uint32_t cur_idx = idx;

        // Insert the new primitive into the ascending t sorted list
        for (int i = 0; i < CHUNK_SIZE; ++i)
        {
            // Swap if the new intersection is closer
            if (payload.buffer[i].tmx > cur_tmx)
            {
                // Store the original buffer info
                tmp_tmx = payload.buffer[i].tmx;
                tmp_idx = payload.buffer[i].idx;
                // Update the current intersection info
                payload.buffer[i].tmx = cur_tmx;
                payload.buffer[i].idx = cur_idx;
                // Swap
                cur_tmx = tmp_tmx;
                cur_idx = tmp_idx;
            }
        }
    }

    // Ignore the intersection to continue traversal
    optixIgnoreIntersection();
}
