#include "hip/hip_runtime.h"
/**
 * @file forward.cu
 * @author xbillowy
 * @brief 
 * @version 0.1
 * @date 2024-08-17
 * 
 * @copyright Copyright (c) 2024
 * 
 */

#define OPTIXU_MATH_DEFINE_IN_NAMESPACE

#include <optix.h>
#include <hip/hip_math_constants.h>

#include "params.h"
#include "auxiliary.h"

// Make the parameters available to the device code
extern "C" {
    __constant__ Params params;
}

// Unpack two 32-bit payload from a 64-bit pointer
static __forceinline__ __device__
void *unpackPointer(uint32_t i0, uint32_t i1) {
    const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}
// Pack a 64-bit pointer from two 32-bit payload
static __forceinline__ __device__
void packPointer(void* ptr, uint32_t& i0, uint32_t& i1) {
    const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}
// Get the payload pointer
template<typename T>
static __forceinline__ __device__ T *getPayload() {
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return reinterpret_cast<T*>(unpackPointer(u0, u1));
}

// Call optixTrace to trace a single ray
__device__ void traceStep(float3 ray_o, float3 ray_d, uint32_t payload_u0, uint32_t payload_u1)
{
    optixTrace(
        params.handle,
        ray_o,
        ray_d,
        0.0f,  // Min intersection distance
        1e16,  // Max intersection distance
        0.0f,  // rayTime, used for motion blur, disable
        OptixVisibilityMask(0xFF),
        OPTIX_RAY_FLAG_NONE,
        0,  // SBT offset
        0,  // SBT stride
        0,  // missSBTIndex
        payload_u0, payload_u1);
}

// Forward method for converting the input spherical harmonics
// coefficients of each Gaussian to a simple RGB color
__device__ glm::vec3 computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs)
{
	// The implementation is loosely based on code for 
	// "Differentiable Point-Based Radiance Fields for 
	// Efficient View Synthesis" by Zhang et al. (2022)
	glm::vec3 pos = means[idx];
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;
	glm::vec3 result = SH_C0 * sh[0];

	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2)
			{
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
	result += 0.5f;
	return glm::max(result, 0.0f);
}


// Compute a 2D-to-2D mapping matrix from world to splat space,
// given a 2D gaussian parameters
__device__ void compute_transmat_uv(
	const glm::vec3 p_orig,
	const glm::vec2 scale,
	float mod,
	const glm::vec4 rot,
	const float* viewmatrix,
    const float3 xyz,
	glm::mat3x4& world2splat,
	float3& normal,
    float2& uv
) {
    // Convert the quaternion and scale vector to matrices
    // * NOTE: R here is the row-major rotation matrix, namely R as in Python,
    // * NOTE: the original quat_to_rotmat(rot) will return the column-major R^T
    // * NOTE: S here is the inverse of the scale matrix
	glm::mat3 R = quat_to_rotmat_transpose(rot);
	glm::mat3 S = scale_to_mat_inverse(scale, mod);
	glm::mat3 L = S * R;
    glm::vec3 T = -1.f * L * p_orig;

	// Compute the world to splat transformation matrix
	world2splat = glm::mat3x4(
        glm::vec4(L[0].x, L[1].x, L[2].x, T.x),
        glm::vec4(L[0].y, L[1].y, L[2].y, T.y),
        glm::vec4(L[0].z, L[1].z, L[2].z, T.z)
	);

    // Compute the normal in world space
	normal = make_float3(L[0].z, L[1].z, L[2].z);

    // Convert the intersection point from world to splat space
    glm::vec3 uv1 = glm::vec4(glm::vec3(xyz.x, xyz.y, xyz.z), 1.0f) * world2splat;
    uv = make_float2(uv1.x, uv1.y);
}


__device__ bool compute_transmat_xy(
	const glm::vec3 p_orig,
	const glm::vec2 scale,
	float mod,
	const glm::vec4 rot,
	const float* projmatrix,
	const int W,
	const int H,
	float2& xy
) {
    // Convert the quaternion and scale vector to matrices
	glm::mat3 R = quat_to_rotmat(rot);  // R
	glm::mat3 S = scale_to_mat(scale, mod);  // S
	glm::mat3 L = R * S;

    // The splat2world matrix
	glm::mat3x4 splat2world = glm::mat3x4(
		glm::vec4(L[0], 0.0),
		glm::vec4(L[1], 0.0),
		glm::vec4(p_orig.x, p_orig.y, p_orig.z, 1)
	);
    // The world2ndc matrix
	glm::mat4 world2ndc = glm::mat4(
		projmatrix[0], projmatrix[4], projmatrix[ 8], projmatrix[12],
		projmatrix[1], projmatrix[5], projmatrix[ 9], projmatrix[13],
		projmatrix[2], projmatrix[6], projmatrix[10], projmatrix[14],
		projmatrix[3], projmatrix[7], projmatrix[11], projmatrix[15]
	);
    // The ndc2pix matrix
	glm::mat3x4 ndc2pix = glm::mat3x4(
		glm::vec4(float(W) / 2.0, 0.0, 0.0, float(W-1) / 2.0),
		glm::vec4(0.0, float(H) / 2.0, 0.0, float(H-1) / 2.0),
		glm::vec4(0.0, 0.0, 0.0, 1.0)
	);

    glm::mat3 T;
    // Compute the final transformation matrix from splat space to pixel space
    // NOTE: The matrix multiplication is performed from right to left
	T = glm::transpose(splat2world) * world2ndc * ndc2pix;  // (3, 4) * (4, 4) * (4, 3) -> (3, 3)

    // Computing the projected center of each 2D Gaussian
    // The projected center of the 2DGS is used to create a low pass filter
	float3 T0 = {T[0][0], T[0][1], T[0][2]};
	float3 T1 = {T[1][0], T[1][1], T[1][2]};
	float3 T3 = {T[2][0], T[2][1], T[2][2]};
	// Compute AABB
	float3 f = {1.0f, 1.0f, -1.0f};
	float a = sumf3(T3 * T3 * f);
    if (a == 0.0) return false;

    float3 c = f / a;
    // Compute the projected center as the center of the AABB
	xy = {sumf3(c * T0 * T3), sumf3(c * T1 * T3)};
    return true;
}


// TODO: throw error if chunk size exceeds
// Core __raygen__ program
extern "C" __global__ void __raygen__ot()
{
    // Lookup current location within the launch grid
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();
    uint32_t tidx = idx.x * dim.y + idx.y;
    // Fetch the ray origin and direction of the current pixel
    float3 ray_om = params.ray_o[tidx];
    float3 ray_dm = params.ray_d[tidx];
    // Store a copy of the original ray origin and direction
    float3 ray_oc = ray_om;
    float3 ray_dc = ray_dm;
    // bool flag=false;
    // if (idx.x == 25 && idx.y == 1000)
    // {
    //     flag = true;
    // }

    // Creat and initialize the ray payload data
    RayPayload payload;
    IntersectionInfo buffer[CHUNK_SIZE];
    for (int i = 0; i < CHUNK_SIZE; i++) buffer[i].tmx = 1e16f;
    payload.buffer = buffer;
    payload.dpt = 0.f;
    payload.cnt = 0.f;

    // Pack the pointer, the values we store the payload pointer in
    uint32_t payload_u0, payload_u1;
    packPointer(&payload, payload_u0, payload_u1);

    // Initialize the volume rendering data
	float3 N = make_float3(0.0f, 0.0f, 0.0f);
    float C[3] = {0.0f};
    float D = 0.0f;
	float W = 0.0f;
	float M1 = {0};
	float M2 = {0};
	float distortion = {0};
	float median_depth = {0};
    float T = 1.0f;
    float test_T = 1.0f;
    float last_dpt = 1e16f;
    int* m_buffer = params.out_attr_uint32 + NUM_CHANNELS_I * tidx + M_BUFFER;

	int contributor = 0;
	int median_contributor = -1;


    // Prepare rendering data
    float dpt = 0.0f;
    float3 xyz = make_float3(0.0f, 0.0f, 0.0f);
    glm::mat3x4 world2splat = glm::mat3x4(0.0f);
    float3 normal = make_float3(0.0f, 0.0f, 0.0f);
    float2 uv = make_float2(0.0f, 0.0f);
    // float2 xy;
    int last_gidx = -1;

    while (1)
    {
        // Actual optixTrace
        traceStep(ray_om, ray_dm, payload_u0, payload_u1);
        // Volume rendering

        for (int i = 0; i < CHUNK_SIZE; i++)
        {
            // Break if the intersection depth is invalid
            if (i >= payload.cnt)
                break;

            // Get the primitive index and Gaussian index
            int pidx = payload.buffer[i].idx;  // intersection primitive index
            int gidx = pidx / 2;  // Gaussian index is half of the primitive index
            // Compute the actual intersection depth and coordinates in world space
            dpt = payload.buffer[i].tmx + payload.dpt;
            xyz = ray_oc + dpt * ray_dc;

            // // Re-initialize payload data
            payload.buffer[i].tmx = 1e16f;
            payload.buffer[i].idx = 0;
            if (gidx == last_gidx)
            {
                continue;
            }
            last_gidx = gidx;
            // last_d = dpt;
            // last_xyz = xyz;

            // Build the world to splat transformation matrix
            // and compute the normal vector
            compute_transmat_uv(params.means3D[gidx], params.scales[gidx],
                                params.scale_modifier, params.rotations[gidx], params.viewmatrix,
                                xyz, world2splat, normal, uv);

            // Adjust the normal vector direction
#if DUAL_VISIABLE
            float3 dir = make_float3(params.means3D[gidx].x - ray_oc.x, params.means3D[gidx].y - ray_oc.y, params.means3D[gidx].z - ray_oc.z);
            // float3 dir = ray_dm;
            float cos = -sumf3(dir * normal);
            if (cos == 0) continue;
            normal = cos > 0 ? normal : -normal;
#endif

            // // Compute the projected center of each 2D Gaussian
            // bool ok = compute_xy(params.means3D[gidx], params.scales[gidx],
            //                      params.scale_modifier, params.rotations[gidx],
            //                      params.projmatrix, params.W, params.H, xy);
            // if (!ok) continue;
            // // TODO (xbillowy): Determine the use of Eq. (11) of 2DGS
            // float rho2d = FilterInvSquare * ((xy.x - idx.x) * (xy.x - idx.x) + (xy.y - idx.y) * (xy.y - idx.y));

            // Get weights
            float rho3d = uv.x * uv.x + uv.y * uv.y;
            float rho2d = rho3d;
            float power = -0.5f * min(rho3d, rho2d);
            if (power > 0.0f)
                continue;

            // Eq. (2) from 3D Gaussian splatting paper.
			// Obtain alpha by multiplying with Gaussian opacity
			// and its exponential falloff from mean.
			// Avoid numerical instabilities (see paper appendix). 
            float alpha = min(0.99f, params.opacities[gidx] * exp(power));
            if (alpha < 1.0f / 255.0f)
                continue;

            test_T = T * (1 - alpha);
            if (test_T < 0.001f)
            {
                break;
            }
            float w = alpha * T;

            // Render colors
            if (params.colors_precomp == nullptr)
            {
                glm::vec3 result = computeColorFromSH(gidx, params.D, params.M,
                                                      params.means3D, *params.campos,
                                                      params.shs);
                C[0] += w * result.x;
                C[1] += w * result.y;
                C[2] += w * result.z;
            }
            else
            {
                for (int ch = 0; ch < 3; ch++)
                    C[ch] += w * params.colors_precomp[ch];
            }
            
			if (T > 0.5) {
				median_depth = dpt;
				// median_weight = w;
				median_contributor = gidx;
			}
            // Render other componments
            atomicAdd(params.accum_gaussian_weights + gidx, w);
            N += w * normal;
            D += w * dpt;
            W += w;
            float A = 1-T;
			float m = dpt;
			distortion += (m * m * A + M2 - 2 * m * M1) * w;
			M1 += m * w;
			M2 += m * m * w;

            // Update transmittence
            T = test_T;

            // Keep track of the current position and last range entry to update this pixel
            m_buffer[contributor] = gidx;
			contributor++;
            // if (flag)
            //     printf("gidx: %d\n", gidx);

        }

        if (test_T < 0.001f || payload.cnt < CHUNK_SIZE)
        {
            break;
        }

        // Re-initialize payload data
        payload.dpt = dpt;
        payload.cnt = 0;
        // Update Ray origin
        ray_om = ray_oc + (payload.dpt + STEP_EPSILON) * ray_dc;
    }

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
    for (int ch = 0; ch < 3; ch++)
        params.out_attr_float32[NUM_CHANNELS_F * tidx + RGB_OFFSET + ch] = 0;
    params.out_attr_float32[NUM_CHANNELS_F * tidx + DEPTH_OFFSET] = D;
    params.out_attr_float32[NUM_CHANNELS_F * tidx + INTENSITY_OFFSET] = C[0] + T * params.background[0];
    params.out_attr_float32[NUM_CHANNELS_F * tidx + RAYHIT_OFFSET] = C[1] + T * params.background[1];
    params.out_attr_float32[NUM_CHANNELS_F * tidx + RAYDROP_OFFSET] = C[2] + T * params.background[2];
    params.out_attr_float32[NUM_CHANNELS_F * tidx + ACCUM_OFFSET] = W;

    params.out_attr_float32[NUM_CHANNELS_F * tidx + NORMAL_OFFSET + 0] = N.x;
    params.out_attr_float32[NUM_CHANNELS_F * tidx + NORMAL_OFFSET + 1] = N.y;
    params.out_attr_float32[NUM_CHANNELS_F * tidx + NORMAL_OFFSET + 2] = N.z;
    
    params.out_attr_float32[NUM_CHANNELS_F * tidx + MIDDEPTH_OFFSET] = median_depth;
    params.out_attr_float32[NUM_CHANNELS_F * tidx + DISTORTION_OFFSET] = distortion;
    params.out_attr_float32[NUM_CHANNELS_F * tidx + FINALT_OFFSET] = T;
    params.out_attr_float32[NUM_CHANNELS_F * tidx + DISDEPTH_OFFSET] = M1;
    params.out_attr_float32[NUM_CHANNELS_F * tidx + DISDEPTH2_OFFSET] = M2;
    
    params.out_attr_uint32[NUM_CHANNELS_I * tidx + MID_CONTRIB_OFFSET] = median_contributor;
    params.out_attr_uint32[NUM_CHANNELS_I * tidx + N_CONTRIB_OFFSET] = contributor;

}


// Core __anyhit__ program
extern "C" __global__ void __anyhit__ot()
{
    // https://forums.developer.nvidia.com/t/some-confusion-on-anyhit-shader-in-optix/223336
    // Get the payload pointer
    RayPayload &payload = *getPayload<RayPayload>();

    // Get the intersection tmax and the primitive index
    float tmx = optixGetRayTmax();
    uint32_t idx = optixGetPrimitiveIndex();

    // Increment the number of intersections
    if (tmx < payload.buffer[CHUNK_SIZE - 1].tmx)
    {
        // Enter this branch means current intersection is closer, we need to update the buffer
        // Increment the counter, the counter only increases when the intersection is closer
        payload.cnt += 1;

        // Temporary variable for swapping
        float tmp_tmx;
        float cur_tmx = tmx;
        uint32_t tmp_idx;
        uint32_t cur_idx = idx;

        // Insert the new primitive into the ascending t sorted list
        for (int i = 0; i < CHUNK_SIZE; ++i)
        {
            // Swap if the new intersection is closer
            if (payload.buffer[i].tmx > cur_tmx)
            {
                // Store the original buffer info
                tmp_tmx = payload.buffer[i].tmx;
                tmp_idx = payload.buffer[i].idx;
                // Update the current intersection info
                payload.buffer[i].tmx = cur_tmx;
                payload.buffer[i].idx = cur_idx;
                // Swap
                cur_tmx = tmp_tmx;
                cur_idx = tmp_idx;
            }
        }
    }

    // Ignore the intersection to continue traversal
    optixIgnoreIntersection();
}
